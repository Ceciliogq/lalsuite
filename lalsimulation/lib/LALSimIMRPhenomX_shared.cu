#define PHENOMXDEBUG 0


#include "LALSimIMRPhenomX_shared.h"


void IMRPhenomX_FillArray(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}



int IMRPhenomX_Frequency_Loop(COMPLEX16FrequencySeries **htilde22,
                     UNUSED  REAL8Sequence *freqs,
                         IMRPhenomXWaveformStruct *pWF,
                         IMRPhenomXAmpCoefficients *pAmp22,
                         IMRPhenomXPhaseCoefficients *pPhase22,
                         UINT4 offset,
                         UINT4 N)
{

  printf("\nInside PhXFreqLoop CUDA file");

  #ifdef __cplusplus
  printf("\nC++\n");
  #else
  printf("\nNo C++\n");
  #endif

  #ifdef __NVCC__
  printf("NVCC defined\n");
  #else
  printf("NVCC NOT defined\n");
  #endif

  #if defined(CUDA)
  printf("CUDA defined\n");
  #else
  printf("CUDA NOT defined\n");
  #endif

  #if defined(LALSIMULATION_CUDA_ENABLED)
  printf("LALSIMULATION_CUDA_ENABLED defined\n");
  #else
  printf("LALSIMULATION_CUDA_ENABLED NOT defined\n");
  #endif

  N = 100; //freqs->length;
  float *x = NULL;// *y = NULL;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged((void**)&x, N*sizeof(float), hipMemAttachGlobal);
  //hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
 // for (int i = 0; i < N; i++) {
 //   x[i] = 1.0f;
 //   y[i] = 2.0f;
 // }

  // Run kernel on 1M elements on the GPU
  //IMRPhenomX_FillArray<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  //hipDeviceSynchronize();

  for (UINT4 idx = 0; idx < N; idx++)
  {
	  /* Reconstruct waveform: h(f) = A(f) * Exp[I phi(f)] */
      ((*htilde22)->data->data)[idx] = 0.*(pWF->amp0 + pAmp22->fAmpMatchIN + pPhase22->C1Int + offset);
  }


  // Free memory
  //hipFree(x);
  //hipFree(y);

  return XLAL_SUCCESS;
}

// int IMRPhenomX_Frequency_Loop(COMPLEX16FrequencySeries **htilde22,
//                          REAL8Sequence *freqs,
//                          IMRPhenomXWaveformStruct *pWF,
//                          IMRPhenomXAmpCoefficients *pAmp22,
//                          IMRPhenomXPhaseCoefficients *pPhase22,
//                          UINT4 offset)
// {
//
//   /* initial_status used to track  */
//   UINT4 initial_status = XLAL_SUCCESS;
//   int status = initial_status;
//
//   /* Linear time and phase shifts so that model peaks near t ~ 0 */
//   REAL8 lina = 0;
//    double linb=IMRPhenomX_TimeShift_22(pPhase22, pWF);
//
//   /* 1/eta is used to re-scale phase */
//   REAL8 inveta    = (1.0 / pWF->eta);
//
//   /*
//       Here we declare explicit REAL8 variables for main loop in order to avoid numerous
//       pointer calls.
//   */
//   //REAL8 MfRef     = pWF->MfRef;
//   REAL8 Msec      = pWF->M_sec;
//
//   REAL8 C1IM      = pPhase22->C1Int;
//   REAL8 C2IM      = pPhase22->C2Int;
//   REAL8 C1RD      = pPhase22->C1MRD;
//   REAL8 C2RD      = pPhase22->C2MRD;
//
//   REAL8 fPhaseIN  = pPhase22->fPhaseMatchIN;
//   REAL8 fPhaseIM  = pPhase22->fPhaseMatchIM;
//   REAL8 fAmpIN    = pAmp22->fAmpMatchIN;
//   REAL8 fAmpIM    = pAmp22->fAmpRDMin;
//
//   REAL8 Amp0      = pWF->amp0 * pWF->ampNorm;
//
//
//
//   for (UINT4 idx = 0; idx < freqs->length; idx++)
//   {
//     double Mf    = Msec * freqs->data[idx];   // Mf is declared locally inside the loop
//     UINT4 jdx    = idx  + offset;             // jdx is declared locally inside the loop
//
//     /* Initialize a struct containing useful powers of Mf */
//     IMRPhenomX_UsefulPowers powers_of_Mf;
//     initial_status     = IMRPhenomX_Initialize_Powers(&powers_of_Mf,Mf);
//     if(initial_status != XLAL_SUCCESS)
//     {
//       status = initial_status;
//       XLALPrintError("IMRPhenomX_Initialize_Powers failed for Mf, initial_status=%d",initial_status);
//     }
//     else
//     {
//       /* Generate amplitude and phase at MfRef */
//       REAL8 amp = 0.0;
//       REAL8 phi = 0.0;
//
//       /* The functions in this routine are inlined to help performance. */
//       /* Construct phase */
//       if(Mf < fPhaseIN)
//       {
//         phi = IMRPhenomX_Inspiral_Phase_22_AnsatzInt(Mf, &powers_of_Mf, pPhase22);
//       }
//       else if(Mf > fPhaseIM)
//       {
//         phi = IMRPhenomX_Ringdown_Phase_22_AnsatzInt(Mf, &powers_of_Mf, pWF, pPhase22) + C1RD + (C2RD * Mf);
//       }
//       else
//       {
//         phi = IMRPhenomX_Intermediate_Phase_22_AnsatzInt(Mf, &powers_of_Mf, pWF, pPhase22) + C1IM + (C2IM * Mf);
//       }
//
// 	  /* Scale phase by 1/eta */
// 	  phi  *= inveta;
//       phi  += linb*Mf + lina + pWF->phifRef;
//
// 	  /* Construct amplitude */
// 	  if(Mf < fAmpIN)
// 	  {
// 		  amp = IMRPhenomX_Inspiral_Amp_22_Ansatz(Mf, &powers_of_Mf, pWF, pAmp22);
// 	  }
// 	  else if(Mf > fAmpIM)
// 	  {
// 		  amp = IMRPhenomX_Ringdown_Amp_22_Ansatz(Mf, pWF, pAmp22);
// 	  }
// 	  else
// 	  {
//         amp = IMRPhenomX_Intermediate_Amp_22_Ansatz(Mf, &powers_of_Mf, pWF, pAmp22);
//       }
//
// 	  /* Reconstruct waveform: h(f) = A(f) * Exp[I phi(f)] */
//       ((*htilde22)->data->data)[jdx] = Amp0 * 0. * powers_of_Mf.m_seven_sixths * amp * cexp(I * phi);
//     }
//   }
//
//     return status;
// }
